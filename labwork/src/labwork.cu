#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            printf("labwork 5 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork5_GPU();
          printf("labwork %d GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
      timer.start();
      labwork.labwork5_GPU_sharedMem();
      printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;

    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {   // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
 int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    #pragma omp parallel for
    for (int j = 0; j < 100; j++) {   // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    printf("my number of of device :  %d\n", numDevices);
    for (int i = 0; i < numDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device #%d\n", i);
        printf("- Name : %s\n", prop.name);
        printf("- Core Info\n");
        printf("    - SPCores : %d\n", getSPcores(prop));
        printf("    - ClockRate : %d\n", prop.clockRate);
        printf("    - MultiProcessor : %d\n", prop.multiProcessorCount);
        printf("    - WarpSize : %d\n", prop.warpSize);
        printf("- Memory Info\n");
        printf("    - ClockRate : %d\n", prop.memoryClockRate);
        printf("    - BusWidth : %d\n", prop.memoryBusWidth);
   }
}


__global__ void grayscale(uchar3 *input, uchar3 *output) {
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
   output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork3_GPU() {
   //(1) 
   uchar3 * devInput;
   uchar3 * devGray;
   int pixelCount = inputImage->width * inputImage->height; 
   outputImage = static_cast<char *>(malloc(pixelCount * 3));
   //(2)
   hipMalloc(&devInput, pixelCount * sizeof (uchar3)); 
   hipMalloc(&devGray, pixelCount * sizeof (uchar3));
   //(3)
   hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
   //(4)
   int dimBlock = 1024;
   int dimGrid = pixelCount / dimBlock;
   grayscale<<<dimGrid, dimBlock>>>(devInput, devGray);
   //(5)

   //(6)
   hipMemcpy(outputImage, devGray, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
   //(7)
   hipFree(devInput);
   hipFree(devGray);
}

__global__ void grayscale2D(uchar3 *input, uchar3 *output, int imageWidth, int imageHeight) {
   int tid = threadIdx.x + blockIdx.x * blockDim.x + (imageWidth*(threadIdx.y+blockIdx.y*blockDim.y));
   if(threadIdx.x > imageHeight) return;
   if(threadIdx.y > imageWidth) return;
   output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
   output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork4_GPU() {
   uchar3 * devInput;
   uchar3 * devGray;
   int pixelCount = inputImage->width * inputImage->height;
   outputImage = static_cast<char *>(malloc(pixelCount * 3));
   hipMalloc(&devInput, pixelCount * sizeof (uchar3));
   hipMalloc(&devGray, pixelCount * sizeof (uchar3));
   hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
   dim3 dimBlock = dim3(32,32);
   dim3 dimGrid = dim3(inputImage->width/32+1, inputImage->height/32+1);
   grayscale2D<<<dimGrid, dimBlock>>>(devInput,
 devGray, inputImage->width, inputImage->height);
   hipMemcpy(outputImage, devGray, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
   hipFree(devInput);
   hipFree(devGray);
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

__global__ void gaussianBlur(uchar3 * input, uchar3 * output, int width, int height) {
   int tidx = threadIdx.x + blockIdx.x * blockDim.x;
   if (tidx >= width) return;
   int tidy = threadIdx.y + blockIdx.y * blockDim.y;
   if (tidy >= height) return;
   int posOut = tidx + tidy * width;
   int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
   int sum = 0;
   int c = 0;
   for ( int x = -3 ; x < 3 ; x++) {
      for ( int y = -3 ; y< 3 ; y++ ) {
     int i = tidx + x;
           int j = tidy + y;
     if (x < 0) continue;
           if (x >= width) continue;
           if (y < 0) continue;
           if (y >= height) continue;
     int tid = j * width + i;
           unsigned char gray = (input[tid].x + input[tid].y + input[tid].x)/3;
           int coefficient = kernel[(y+3) * 7 + x + 3];
           sum = sum + gray * coefficient;
           c += coefficient;    
      }
   }
   sum /= c;
   output[posOut].x = output[posOut].y = output[posOut].z = sum;
}


void Labwork::labwork5_GPU() {
   uchar3 * devInput;
   uchar3 * devBlur;
   int pixelCount = inputImage->width * inputImage->height;
   outputImage = static_cast<char *>(malloc(pixelCount * 3));
   hipMalloc(&devInput, pixelCount * sizeof (uchar3));
   hipMalloc(&devBlur, pixelCount * sizeof (uchar3));
   hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
   
   dim3 dimBlock = dim3(32,32);
   dim3 dimGrid = dim3(inputImage->width/32+1, inputImage->height/32+1);
   gaussianBlur<<<dimGrid, dimBlock>>>(devInput, devBlur, inputImage->width, inputImage->height);
   hipMemcpy(outputImage, devBlur, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
   hipFree(devInput);
   hipFree(devBlur);
}


__global__ void gaussianBlurSharedMem(uchar3 * input, uchar3 * output, int width, int height, int kernel[]) {
   int tidx = threadIdx.x + blockIdx.x * blockDim.x;
   if (tidx >= width) return;
   int tidy = threadIdx.y + blockIdx.y * blockDim.y;
   if (tidy >= height) return;
   int posOut = tidx + tidy * width;
   __shared__ int skernel[49];
   if (posOut < 49) {
    skernel[posOut] = kernel[posOut];
   }
   __syncthreads();

   int sum = 0;
   int c = 0;
   for ( int x = -3 ; x < 3 ; x++) {
      for ( int y = -3 ; y< 3 ; y++ ) {
     int i = tidx + x;
           int j = tidy + y;
     if (x < 0) continue;
           if (x >= width) continue;
           if (y < 0) continue;
           if (y >= height) continue;
     int tid = j * width + i;
           unsigned char gray = (input[tid].x + input[tid].y + input[tid].x)/3;
           int coefficient = skernel[(y+3) * 7 + x + 3];
           sum = sum + gray * coefficient;
           c += coefficient;    
      }
   }
   sum /= c;
   output[posOut].x = output[posOut].y = output[posOut].z = sum;
}


void Labwork::labwork5_GPU_sharedMem() {
   uchar3 * devInput;
   uchar3 * devBlur;
   int pixelCount = inputImage->width * inputImage->height;
   outputImage = static_cast<char *>(malloc(pixelCount * 3));
   hipMalloc(&devInput, pixelCount * sizeof (uchar3));
   hipMalloc(&devBlur, pixelCount * sizeof (uchar3));
   hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
   int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
   dim3 dimBlock = dim3(32,32);
   dim3 dimGrid = dim3(inputImage->width/32+1, inputImage->height/32+1);
   gaussianBlurSharedMem<<<dimGrid, dimBlock>>>(devInput, devBlur, inputImage->width, inputImage->height, kernel);
   hipMemcpy(outputImage, devBlur, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
   hipFree(devInput);
   hipFree(devBlur);
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}