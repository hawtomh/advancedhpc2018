#include "hip/hip_runtime.h"
 #include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv)
{
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2)
    {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 )
    {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum)
    {
    case 1:
        labwork.labwork1_CPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork2-cpu-out.jpg");
        timer.start();
        labwork.labwork1_OpenMP();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork2-openmp-out.jpg");
        break;
    case 2:
        labwork.labwork2_GPU();
        break;
    case 3:
        labwork.labwork3_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork3-gpu-out.jpg");
        break;
    case 4:
        labwork.labwork4_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork4-gpu-out.jpg");
        break;
    case 5:
        labwork.labwork5_CPU();
        labwork.saveOutputImage("labwork5-cpu-out.jpg");
        printf("labwork 5 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        timer.start();
        labwork.labwork5_GPU();
        printf("labwork %d GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        timer.start();
        labwork.labwork5_GPU_sharedMem();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork5-gpu-out.jpg");
        break;
    case 6:
      

        labwork.loadInputImage2(argv[3]);
        timer.getElapsedTimeInMilliSec();
        timer.start();
        labwork.labwork6_GPU(1, 127);
        printf("labwork %d binari ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
                labwork.saveOutputImage("labwork6-bin-gpu-out.jpg");
                timer.start();
        labwork.labwork6_GPU(2, -10);
        printf("labwork %d brightness ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
                labwork.saveOutputImage("labwork6-bright-gpu-out.jpg");
                timer.start();
        labwork.labwork6_GPU(3, 0.50);
        printf("labwork %d blending ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork6-blend-gpu-out.jpg");
        break;
    case 7:
        labwork.labwork7_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork7-gpu-out.jpg");
        break;
    case 8:
        labwork.labwork8_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork8-gpu-out.jpg");
        break;
    case 9:
        labwork.labwork9_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork9-gpu-out.jpg");
        break;
    case 10:
        labwork.labwork10_GPU();
        printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
        labwork.saveOutputImage("labwork10-gpu-out.jpg");
        break;
    }
    printf("Program ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName)
{
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::loadInputImage2(std::string inputFileName)
{
    inputImage2 = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName)
{
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

/********************
 *
 *    Labwork 1
 *
 ********************/

void Labwork::labwork1_CPU()
{
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++)     // let's do it 100 times, otherwise it's too fast!
    {
        for (int i = 0; i < pixelCount; i++)
        {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP()
{
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    #pragma omp parallel for
    for (int j = 0; j < 100; j++)     // let's do it 100 times, otherwise it's too fast!
    {
        for (int i = 0; i < pixelCount; i++)
        {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

/********************
 *
 *    Labwork 2
 *
 ********************/

int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major)
    {
    case 2: // Fermi
        if (devProp.minor == 1) cores = mp * 48;
        else cores = mp * 32;
        break;
    case 3: // Kepler
        cores = mp * 192;
        break;
    case 5: // Maxwell
        cores = mp * 128;
        break;
    case 6: // Pascal
        if (devProp.minor == 1) cores = mp * 128;
        else if (devProp.minor == 0) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    default:
        printf("Unknown device type\n");
        break;
    }
    return cores;
}

void Labwork::labwork2_GPU()
{
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    printf("my number of of device :  %d\n", numDevices);
    for (int i = 0; i < numDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device #%d\n", i);
        printf("- Name : %s\n", prop.name);
        printf("- Core Info\n");
        printf("    - SPCores : %d\n", getSPcores(prop));
        printf("    - ClockRate : %d\n", prop.clockRate);
        printf("    - MultiProcessor : %d\n", prop.multiProcessorCount);
        printf("    - WarpSize : %d\n", prop.warpSize);
        printf("- Memory Info\n");
        printf("    - ClockRate : %d\n", prop.memoryClockRate);
        printf("    - BusWidth : %d\n", prop.memoryBusWidth);
    }
}

/********************
 *
 *    Labwork 3
 *
 ********************/


__global__ void grayscale(uchar3 *input, uchar3 *output, int width, int height)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= width * height) return;
    unsigned char g = (input[tid].x + input[tid].y + input[tid].z) / 3;
    output[tid].z = output[tid].y = output[tid].x = g;
}

void Labwork::labwork3_GPU()
{
  // Host data
  int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    hipError_t r;
    
    // Device data
    uchar3 *devInput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount * sizeof (uchar3));
    hipMalloc(&devGray, pixelCount * sizeof (uchar3));
    
    // Transfert host data to device
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
    
    // Device process the data
    int dimBlock = 1024;
    int nbBlock = ceil(pixelCount / dimBlock); 
    grayscale <<< nbBlock, dimBlock>>>(devInput, devGray, inputImage->width, inputImage->height);

  // Copy back the dat from the device to the host
    hipMemcpy(outputImage, devGray, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devInput);
    hipFree(devGray);
    
    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 4
 *
 ********************/

__global__ void grayscale2D(uchar3 *input, uchar3 *output, int width, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
    int tid = tidx + tidy * width;
    unsigned char g = (input[tid].x + input[tid].y + input[tid].z) / 3;
    output[tid].z = output[tid].y = output[tid].x = g;
}

void Labwork::labwork4_GPU()
{
  // Host data
  int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    hipError_t r;
    
    // Device data
    uchar3 *devInput;
    uchar3 *devGray;
    hipMalloc(&devInput, pixelCount * sizeof (uchar3));
    hipMalloc(&devGray, pixelCount * sizeof (uchar3));
    
    // Transfert host data to device
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
    
    // Device process the data
    dim3 dimBlock2d = dim3(32, 32); // 32^2 = 1024 = number of threads per block
  dim3 nbBlock2d = dim3(ceil((double)inputImage->width/dimBlock2d.x), ceil((double)inputImage->height/dimBlock2d.y));
    grayscale2D <<< nbBlock2d, dimBlock2d>>>(devInput, devGray, inputImage->width, inputImage->height);
    
  // Copy back the dat from the device to the host
    hipMemcpy(outputImage, devGray, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devInput);
    hipFree(devGray);
    
    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 5
 *
 ********************/

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU()
{
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0
                   };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char *) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++)
    {
        for (int col = 0; col < inputImage->width; col++)
        {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++)
            {
                for (int x = -3; x <= 3; x++)
                {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2]) / 3;
                    int coefficient = kernel[(y + 3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

__global__ void gaussianBlur(uchar3 *input, uchar3 *output, int width, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
    int posOut = tidx + tidy * width;
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0
                   };
    int sum = 0;
    int c = 0;
    for ( int x = -3 ; x <= 3 ; x++)
    {
        for ( int y = -3 ; y <= 3 ; y++ )
        {
            int i = tidx + x;
            int j = tidy + y;
            if (i < 0) continue;
            if (i >= width) continue;
            if (j < 0) continue;
            if (j >= height) continue;
            int tid = j * width + i;
            unsigned char gray = (input[tid].x + input[tid].y + input[tid].x) / 3;
            int coefficient = kernel[(y + 3) * 7 + x + 3];
            sum = sum + gray * coefficient;
            c += coefficient;
        }
    }
    sum /= c;
    output[posOut].x = output[posOut].y = output[posOut].z = sum;
}


void Labwork::labwork5_GPU()
{
  // Host data
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    hipError_t r;
    
  // Device data    
    uchar3 *devInput;
    uchar3 *devBlur;
    hipMalloc(&devInput, pixelCount * sizeof (uchar3));
    hipMalloc(&devBlur, pixelCount * sizeof (uchar3));
    
    // Transfert host data to device
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);

    // Device process the data
    dim3 dimBlock2d = dim3(32, 32);
  dim3 nbBlock2d = dim3(ceil((double)inputImage->width/dimBlock2d.x), ceil((double)inputImage->height/dimBlock2d.y));
    gaussianBlur <<< nbBlock2d, dimBlock2d>>>(devInput, devBlur, inputImage->width, inputImage->height);
    
  // Copy back the dat from the device to the host
    hipMemcpy(outputImage, devBlur, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devInput);
    hipFree(devBlur);

    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}


__global__ void gaussianBlurSharedMem(uchar3 *input, uchar3 *output, int width, int height, int *kernel )
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
    int posOut = threadIdx.x + threadIdx.y * blockDim.x;

    __shared__ int skernel[49];
    if (posOut < 49)
    {
        skernel[posOut] = kernel[posOut];
    }
    __syncthreads(); // Only the 49 first threads do this, then sync

    int sum = 0;
    int c = 0;
    for ( int x = -3 ; x < 3 ; x++)
    {
        for ( int y = -3 ; y < 3 ; y++ )
        {
            int i = tidx + x;
            int j = tidy + y;
            if (i < 0) continue;
            if (i >= width) continue;
            if (j < 0) continue;
            if (j >= height) continue;
            int tid = j * width + i;
            unsigned char gray = (input[tid].x + input[tid].y + input[tid].x) / 3;
            int coefficient = skernel[(y + 3) * 7 + x + 3];
            sum = sum + gray * coefficient;
            c += coefficient;
        }
    }
    sum /= c;
    posOut = tidx + tidy * width;
    output[posOut].x = output[posOut].y = output[posOut].z = sum;
}


void Labwork::labwork5_GPU_sharedMem()
{
  // Host data
  int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,
                     0, 3, 13, 22, 13, 3, 0,
                     1, 13, 59, 97, 59, 13, 1,
                     2, 22, 97, 159, 97, 22, 2,
                     1, 13, 59, 97, 59, 13, 1,
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0
                   };
    hipError_t r;
    
  // Device data
    uchar3 *devInput;
    uchar3 *devBlur;
    int *devKernel;
    hipMalloc(&devInput, pixelCount * sizeof (uchar3));
    hipMalloc(&devBlur, pixelCount * sizeof (uchar3));
    hipMalloc(&devKernel, sizeof(kernel));
    
    // Transfert host data to device
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice); 
    hipMemcpy(devKernel, kernel, sizeof(kernel), hipMemcpyHostToDevice);
    
    // Device process the data
  dim3 dimBlock2d = dim3(32, 32);
  dim3 nbBlock2d = dim3(ceil((double)inputImage->width/dimBlock2d.x), ceil((double)inputImage->height/dimBlock2d.y));
    gaussianBlurSharedMem <<<nbBlock2d, dimBlock2d>>>(devInput, devBlur, inputImage->width, inputImage->height, devKernel);
    
    // Copy back the data from Device to the host
    hipMemcpy(outputImage, devBlur, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devInput);
    hipFree(devBlur);
    hipFree(devKernel);
    
    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 6
 *
 ********************/

__global__ void binari(uchar3 *input, uchar3 *output, int width, int height,  int threshold)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= width * height) return;
    unsigned int g = (int) ( input[tid].x / threshold ) * 255;
    output[tid].z = output[tid].y = output[tid].x = (char) g;
}

__global__ void brightness(uchar3 *input, uchar3 *output, int width, int height,  int brightnessCoef)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= width * height) return;
    unsigned char g = min(max(input[tid].x + brightnessCoef, 0), 255);  // Stay in [0;255] without BAD UGLY IF
    output[tid].x = output[tid].y = output[tid].z = g;
}

__global__ void blending(uchar3 *input1, uchar3 *input2, uchar3 *output, double percent, int width, int height, int width2, int height2)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= width * height || tid >= width2 * height2) return;
    output[tid].x = (percent * (double) input1[tid].x) + ((1.0 - percent) * (double) input2[tid].x);
    output[tid].y = (percent * (double) input1[tid].y) + ((1.0 - percent) * (double) input2[tid].y);
    output[tid].z = (percent * (double) input1[tid].z) + ((1.0 - percent) * (double) input2[tid].z);
}   


void Labwork::labwork6_GPU(int method, int detailArg)
{
  // Host data
  int pixelCount = inputImage->width * inputImage->height;
  int pixelCount2 = inputImage2->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    hipError_t r;
    
    // Device Data
    uchar3 *devInput;
    uchar3 *devGray;
    uchar3 *devOutput;
    uchar3 *devInput2;
    hipMalloc(&devInput, pixelCount * sizeof (uchar3));
    hipMalloc(&devOutput, pixelCount * sizeof (uchar3));
    hipMalloc(&devGray, pixelCount * sizeof (uchar3));
    hipMalloc(&devInput2, pixelCount2 * sizeof (uchar3));

    
    // Transfert data from host to device
    hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof (uchar3), hipMemcpyHostToDevice);
    hipMemcpy(devInput2, inputImage2->buffer, pixelCount2 * sizeof (uchar3), hipMemcpyHostToDevice);
    
    // Device process the data
    int dimBlock = 1024;
    int dimGrid = ceil(pixelCount / dimBlock);
    if (method == 1) // Launch Binari kernel
    {
      grayscale<<<dimGrid, dimBlock>>>(devInput, devGray, inputImage->width, inputImage->height);
      binari<<<dimGrid, dimBlock>>>(devGray, devOutput, inputImage->width, inputImage->height, detailArg);
    }
    else if(method == 2) // Launch Brightness kernel
    {
        grayscale<<<dimGrid, dimBlock>>>(devInput, devGray, inputImage->width, inputImage->height);
      brightness<<<dimGrid, dimBlock>>>(devGray, devOutput, inputImage->width, inputImage->height,detailArg);
    }
    else if (method == 3) // Launch Blending kernel
    {
      blending<<<dimGrid, dimBlock>>>(devInput, devInput2, devOutput, 0.50, inputImage->width, inputImage->height, inputImage2->width, inputImage2->height);
  }
  
  // Copy back the data from device to the host
    hipMemcpy(outputImage, devOutput, pixelCount * sizeof (uchar3), hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devInput);
    hipFree(devGray);
    hipFree(devOutput);
    hipFree(devInput2);
    
  // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }

}

/********************
 *
 *    Labwork 7
 *
 ********************/

__global__ void uchar3ToInt(uchar3 * input, int * outputMin, int * outputMax, int width, int height) 
{
  // Get the global thread ID with half of block
  int tid = threadIdx.x + blockIdx.x * blockDim.x * 2; 
  if(tid+blockDim.x >= width * height) return;
  // Store the min/max from threadId of the current block and threadId of the "next" block
  outputMin[threadIdx.x + blockIdx.x * blockDim.x] = min(input[tid].x, input[tid + blockDim.x].x);
  outputMax[threadIdx.x + blockIdx.x * blockDim.x] = max(input[tid].x, input[tid + blockDim.x].x);
}

__global__ void reduceToMin(int * minTab) 
{
  extern __shared__ int cache[]; // Size of blockDim
  
  unsigned int localTid = threadIdx.x; // Local thread ID of the current block
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x * 2; // Global thread ID multiply by two because there is half of block
  
  cache[localTid] = min(minTab[tid], minTab[tid + blockDim.x]); // Store the min between the pair and the impair block
  
  __syncthreads(); // Synchronize to be sure cache[] is complete
  
  // REDUCE
  for (int r = blockDim.x  / 2; r > 0; r /= 2) 
  {
  
    if (localTid < r)
    {
      cache[localTid] = min(cache[localTid], cache[localTid + r]);
    }
    
    __syncthreads(); // Synchronize between each reduce step
  }
  
  // Store result
  if (localTid == 0)
  {
    minTab[blockIdx.x] = cache[0]; // The reduction put the minimum between all thread's data of one block
  }
}

__global__ void reduceToMax(int * maxTab) 
{
  extern __shared__ int cache[]; // Size of blockDim
  
  unsigned int localTid = threadIdx.x; // Local thread ID of the current block
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x * 2; // Global thread ID multiply by two because there is half of block
  cache[localTid] = max(maxTab[tid], maxTab[tid + blockDim.x]); // Store the max between the pair and the impair block
  
  __syncthreads(); // Synchronize to be sure cache[] is complete
  
  // REDUCE
  for (int r = blockDim.x  / 2; r > 0; r /= 2) 
  {
    if (localTid < r)
    {
      cache[localTid] = max(cache[localTid], cache[localTid + r]);
    }
    
    __syncthreads(); // Synchronize between each reduce step
  }
  
  // Store result
  if (localTid == 0)
  {
    maxTab[blockIdx.x] = cache[0]; // The reduction put the maximum between all thread's data of one block
  }
}

__global__ void grayscaleStretch(uchar3 * input, uchar3 * output, int * minG, int * maxG, int width, int height)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= width * height) return;
    unsigned char g = (double(input[tid].x - minG[0]) / double(maxG[0] - minG[0])) * 255;
    output[tid].x = output[tid].y = output[tid].z = g;
}

void Labwork::labwork7_GPU()
{
    // Host data
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    int dimBlock = 1024;
    int nbBlock = ceil((double)pixelCount / dimBlock);
    int cacheSize = dimBlock * sizeof(int);
    hipError_t r;
  
    // Device data
    uchar3 *devOutput;
    uchar3 *devImage;
    uchar3 *devGray;
    int *devMin;
    int *devMax;
    hipMalloc(&devOutput, pixelCount * sizeof(uchar3));  
    hipMalloc(&devImage, pixelCount * sizeof(uchar3));
    hipMalloc(&devGray, pixelCount * sizeof(uchar3));
    hipMalloc(&devMin, pixelCount * sizeof(int) / 2);
    hipMalloc(&devMax, pixelCount * sizeof(int) / 2);  
  
    hipMemcpy(devImage, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
  
    // Ask device to process data
    int currentNbBlock = nbBlock/2;
    // (1) Convert RGB to gray
    grayscale <<<nbBlock, dimBlock>>>(devImage, devGray, inputImage->width, inputImage->height);
    // (2) Extract min and max arrays by comparison with half of bock
    uchar3ToInt <<<currentNbBlock, dimBlock>>>(devGray, devMin, devMax, inputImage->width, inputImage->height);
    // (3) Apply REDUCE on min&max arrays until there are more than 1024 entries
    do
    {
      currentNbBlock /= 2;
      reduceToMin<<<currentNbBlock, dimBlock, cacheSize>>>(devMin);
      reduceToMax<<<currentNbBlock, dimBlock, cacheSize>>>(devMax);
    }while(currentNbBlock > 1);
    // (4) Stretch the gray image
    grayscaleStretch<<<nbBlock, dimBlock>>>(devGray, devOutput, devMin, devMax,  inputImage->width, inputImage->height);
  
    // Return data to host
    hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(devImage);
    hipFree(devGray);
    hipFree(devOutput);
    hipFree(devMin);
    hipFree(devMax);
    
    // Get the errors
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 8
 *
 ********************/
 
struct hsv
{
  double *h;
  double *s;
  double *v;
};

__global__ void RGB2HSV(uchar3 *input, hsv output, int width, int height)
{
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
  int tid = tidx + tidy * width;
    
    uchar3 rgb = input[tid];
    int maxRgb = max(rgb.x, max(rgb.y,rgb.z));
    
    double delta = (maxRgb - min(rgb.x, min(rgb.y,rgb.z))) / 255.0;
    double maxRgbReduced = maxRgb / 255.0;
    double R = rgb.x / 255.0;
    double G = rgb.y / 255.0;
    double B = rgb.z / 255.0;
    
    // Define the V
    output.v[tid] = maxRgbReduced;
    
    // Define the S
    if(maxRgb != 0)
    {
      output.s[tid] = delta / maxRgbReduced;
    } 
    else 
    {
      output.s[tid] = 0;
    }
    
    // Define the H
    if (delta == 0)
    {
      output.h[tid] = 0;
    }
    else if (maxRgb == rgb.x)
    {
      output.h[tid] = 60 * fmodf(((G - B) / delta), 6.0);
    }
    else if (maxRgb == rgb.y)
    {
      output.h[tid] = 60 * (((B - R) / delta) + 2);
    }
    else if (maxRgb == rgb.z)
    {
    output.h[tid] = 60 * (((R - G) / delta) + 4);
    }
}

__global__ void HSV2RGB(hsv input, uchar3 *output, int width, int height)
{
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
  int tid = tidx + tidy * width;

    // Store input data in the local memory to reduce the number of access
    double h = input.h[tid];
    double s = input.s[tid];
    double v = input.v[tid];
  
  double d = h / 60;
  int hi = (int) fmodf(d, 6);
  double f = d - hi;
  double l = v * (1 - s);
  double m = v * (1 - f * s);
  double n = v * (1 - (1 - f) * s);
  
  l = floor(l * 255 + 0.5); // Better approximation, 2.1 become 2 and 2.6 become 3 (because round method can't be used in __global__)
  m = floor(m * 255 + 0.5);
  n = floor(n * 255 + 0.5);
  int V = floor(v * 255 + 0.5); 
  
  if (h < 60)
  {
    output[tid].x = V;
    output[tid].y = n;
    output[tid].z = l;
  }
  else if (h < 120)
  {
    output[tid].x = m;
    output[tid].y = V;
    output[tid].z = l;
    return;
  }
  else if (h < 180)
  {
    output[tid].x = l;
    output[tid].y = V;
    output[tid].z = n;
  }
  else if (h < 240)
  {
    output[tid].x = l;
    output[tid].y = m;
    output[tid].z = V;
  }
  else if (h < 300)
  {
    output[tid].x = n;
    output[tid].y = l;
    output[tid].z = V;
  }
  else
  {
    output[tid].x = V;
    output[tid].y = l;
    output[tid].z = m;
  }
}

void Labwork::labwork8_GPU()
{
  // Host data
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    dim3 dimBlock2d = dim3(32,32);
  dim3 nbBlock2d = dim3(ceil((double)inputImage->width/dimBlock2d.x), ceil((double)inputImage->height/dimBlock2d.y));
  hipError_t r;
  
    // Device data
    uchar3 *devRGB;
    uchar3 *devOutput;
    hsv devHSV;
    hipMalloc(&devRGB, pixelCount * sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount * sizeof(uchar3));  
    hipMalloc((void**)&devHSV.h, pixelCount * sizeof(double));
    hipMalloc((void**)&devHSV.s, pixelCount * sizeof(double));
    hipMalloc((void**)&devHSV.v, pixelCount * sizeof(double));
    
    // Prepare data
    hipMemcpy(devRGB, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);

  // Process data
    RGB2HSV<<<nbBlock2d, dimBlock2d>>>(devRGB, devHSV, inputImage->width, inputImage->height);
    HSV2RGB<<<nbBlock2d, dimBlock2d>>>(devHSV, devOutput, inputImage->width, inputImage->height);

  // Get back the data
    hipMemcpy(outputImage, devOutput, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);    
    
    // Free the device
    hipFree(devRGB);
    hipFree(devHSV.h);
    hipFree(devHSV.s);
    hipFree(devHSV.v);
    hipFree(devOutput);
    
    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 9
 *
 ********************/
 
struct histogram {
  int h[256];
};
 
__global__ void uchar3ToTabOfHisto(uchar3 *input, histogram *histo,int localHistoSize, int width, int height)
{
  int cache[256] = {0}; // Very slow :(
  
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
  for (int i = 0; i < localHistoSize; i ++)
  {
    if (tid * localHistoSize + i >= width * height) continue;
    cache[input[tid * localHistoSize + i].x]++;
  }

  for (int i = 0; i < 256; i++)
  {
    histo[tid].h[i] = cache[i];
  }
}

__global__ void reduceHisto(histogram *histo, int nbHistoMax)
{
  unsigned int localTid = threadIdx.x;
  unsigned int tid = blockIdx.x;
  unsigned int halfOfNbHisto = ceil((double)nbHistoMax/2); // Simple reduction by two
  if (tid + halfOfNbHisto >= nbHistoMax) return;
  histo[tid].h[localTid] += histo[tid + halfOfNbHisto].h[localTid];
}

__global__ void computeCDF(histogram *histo,int pixelCount)
{
  int minCdf = 0;
  int cumul = 0;
  
  for (int i = 0; i < 256; i++)
  {
    if (minCdf == 0)
    {
      minCdf = histo[0].h[i];
    }
    cumul += histo[0].h[i];
    histo[0].h[i] = round((double) (cumul - minCdf) / (pixelCount - minCdf) * 255.0);
  }
}

__global__ void equalizer(uchar3 *input, uchar3 *output, histogram *histo, int width, int height)
{ 
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= width * height) return;
  output[tid].x = output[tid].y = output[tid].z = histo[0].h[input[tid].x];
}

void Labwork::labwork9_GPU()
{
  // Host data
    int pixelCount = inputImage->width * inputImage->height;
  outputImage = static_cast<char *>(malloc(pixelCount * 3));
    int dimBlock = 1024;
    int localHistoSize = 1024;
    int nbBlock = ceil((double)pixelCount / dimBlock / localHistoSize); // local histo of one thread will considere 1024px
    hipError_t r;
    
    int currentDimBlock = 256;
    int currentNbHisto = ceil((double)pixelCount/localHistoSize);
    int currentNbBlock = currentNbHisto;
  
  // Device data
  uchar3 *devInput;
  uchar3 *devGray;
  uchar3 *devOutput;
  histogram *devHisto;
  hipMalloc(&devInput, pixelCount * sizeof(uchar3));
  hipMalloc(&devGray, pixelCount * sizeof(uchar3));
  hipMalloc(&devOutput, pixelCount * sizeof(uchar3));
  hipMalloc(&devHisto, currentNbHisto * sizeof(histogram));
  
  // Transfert host data to device  
  hipMemcpy(devInput, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
  
  // Process data
  grayscale <<<nbBlock * localHistoSize, dimBlock>>>(devInput, devGray, inputImage->width, inputImage->height);
    uchar3ToTabOfHisto <<<nbBlock, dimBlock>>>(devGray, devHisto,localHistoSize, inputImage->width, inputImage->height);
    
    // Divide by two the number of histogram at each iteration
    do
    {
      currentNbHisto = currentNbBlock;
      currentNbBlock = ceil((double) currentNbBlock / 2);
      reduceHisto<<<currentNbBlock, currentDimBlock>>>(devHisto, currentNbHisto);
    }while(currentNbBlock > 1);
    computeCDF<<<1,1>>>(devHisto, pixelCount); // Just one thread to transform the original histogram to an cumulated equalized histogram
    equalizer<<<nbBlock * localHistoSize, dimBlock>>>(devGray, devOutput, devHisto,  inputImage->width, inputImage->height);
  
    // Return data to host
    hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
  
  // Free the device
  hipFree(devInput);
  hipFree(devGray);
  hipFree(devOutput);
  
  // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}

/********************
 *
 *    Labwork 10
 *
 ********************/
 
__global__ void kuwaharaFilter(uchar3 * input, hsv HSV,uchar3 *output, int regionSize, int width, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx >= width) return;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidy >= height) return;
    int tid = tidx + tidy * width;
    
    double regionAverage[4] = {0}; // Store the average of V of each region
    int regionPixelCount[4] = {0}; // Store the number of pixel for each region, to process correctly when a region is cut by image' border
    
    int regionAverageColorR[4] = {0}; // Store the average of R of each region
    int regionAverageColorG[4] = {0}; // Store the average of G of each region
    int regionAverageColorB[4] = {0}; // Store the average of B of each region
    
    double regionSd[4] = {0.0}; // Store the standard derivation of each region
    
    // Compute Average of each region for R, G, B and V
    for (int x = 1 - regionSize; x <= regionSize - 1; x++)
    {
      for (int y = 1 - regionSize; y <= regionSize - 1; y++)
      {
        int i = tidx + x;
            int j = tidy + y;
            if (i < 0) continue;
            if (i >= width) continue;
            if (j < 0) continue;
            if (j >= height) continue;
            int currentPos =j * width + i;

            // Top-Left region
            if (x <= 0 && y <= 0)
            {
        regionAverage[0] += HSV.v[currentPos];
        regionAverageColorR[0] += input[currentPos].x;
        regionAverageColorG[0] += input[currentPos].y;
        regionAverageColorB[0] += input[currentPos].z;
        regionPixelCount[0] ++;             
            }
            // Top-Right region
            if (x >= 0 && y <= 0)
            {
              regionAverage[1] += HSV.v[currentPos];
        regionAverageColorR[1] += input[currentPos].x;
        regionAverageColorG[1] += input[currentPos].y;
        regionAverageColorB[1] += input[currentPos].z;
        regionPixelCount[1] ++;
            }
            // Bottom-Left region
            if (x <= 0 && y >= 0)
            {
              regionAverage[2] += HSV.v[currentPos];
        regionAverageColorR[2] += input[currentPos].x;
        regionAverageColorG[2] += input[currentPos].y;
        regionAverageColorB[2] += input[currentPos].z;
        regionPixelCount[2] ++;
            }
            // Bottom-right region
            if (x >= 0 && y >= 0)
            {
              regionAverage[3] += HSV.v[currentPos];
        regionAverageColorR[3] += input[currentPos].x;
        regionAverageColorG[3] += input[currentPos].y;
        regionAverageColorB[3] += input[currentPos].z;
        regionPixelCount[3] ++;
            }
      }
    }
    
    for (int i = 0; i < 4; i ++)
    {
      regionAverage[i] /= regionPixelCount[i];
    regionAverageColorR[i] /= regionPixelCount[i];
    regionAverageColorG[i] /= regionPixelCount[i];
    regionAverageColorB[i] /= regionPixelCount[i];
    } 
  
  // Compute the Standard derivation for each region
  for (int x = 1 - regionSize; x <= regionSize - 1; x++)
    {
      for (int y = 1 - regionSize; y <= regionSize - 1; y++)
      {
        int i = tidx + x;
            int j = tidy + y;
            if (i < 0) continue;
            if (i >= width) continue;
            if (j < 0) continue;
            if (j >= height) continue;
            int currentPos = j * width + i;
            
            // Top-Left region
            if (x <= 0 && y <= 0)
            {
        regionSd[0] += pow((HSV.v[currentPos] - regionAverage[0]), 2.0);
            }
            // Top-Right region
            if (x >= 0 && y <= 0)
            {
              regionSd[1] += pow((HSV.v[currentPos] - regionAverage[1]), 2.0);
            }
            // Bottom-Left region
            if (x <= 0 && y >= 0)
            {
              regionSd[2] += pow((HSV.v[currentPos] - regionAverage[2]), 2.0);
            }
            // Bottom-right region
            if (x >= 0 && y >= 0)
            {
              regionSd[3] += pow((HSV.v[currentPos] - regionAverage[3]), 2.0);
            }
      }
    }
    
    for (int i = 0; i < 4; i ++)
    {
      regionSd[i] = sqrt(regionSd[i] / regionPixelCount[i]);
    }
    
    // Output is define by the lowest SD computed
    if(regionSd[0] <= regionSd[1] && regionSd[0] <= regionSd[2] && regionSd[0] <= regionSd[3])
    {
      output[tid].x = regionAverageColorR[0];
      output[tid].y = regionAverageColorG[0];
      output[tid].z = regionAverageColorB[0];
    }
    else if(regionSd[1] <= regionSd[2] && regionSd[1] <= regionSd[3])
    {
      output[tid].x = regionAverageColorR[1];
      output[tid].y = regionAverageColorG[1];
      output[tid].z = regionAverageColorB[1];
    }
    else if(regionSd[2] <= regionSd[3])
    {
      output[tid].x = regionAverageColorR[2];
      output[tid].y = regionAverageColorG[2];
      output[tid].z = regionAverageColorB[2];
    }
    else
    {
      output[tid].x = regionAverageColorR[3];
      output[tid].y = regionAverageColorG[3];
      output[tid].z = regionAverageColorB[3];
    }
}

void Labwork::labwork10_GPU()
{
  // Host data
  int pixelCount = inputImage->width * inputImage->height;
  outputImage = static_cast<char *>(malloc(pixelCount * 3));
    dim3 dimBlock2d = dim3(16,16);
  dim3 nbBlock2d = dim3(ceil((double)inputImage->width/dimBlock2d.x), ceil((double)inputImage->height/dimBlock2d.y));
    hipError_t r;
    
    // Device data
    int regionSize = 3;
    uchar3 *devRGB;
    uchar3 *devOutput;
    hsv devHSV;
    hipMalloc(&devRGB, pixelCount * sizeof(uchar3));
    hipMalloc(&devOutput, pixelCount * sizeof(uchar3));  
    hipMalloc((void**)&devHSV.h, pixelCount * sizeof(double));
    hipMalloc((void**)&devHSV.s, pixelCount * sizeof(double));
    hipMalloc((void**)&devHSV.v, pixelCount * sizeof(double));
    
    hipMemcpy(devRGB, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
    
    // Process data
    RGB2HSV<<<nbBlock2d, dimBlock2d>>>(devRGB, devHSV, inputImage->width, inputImage->height);
    kuwaharaFilter<<<nbBlock2d, dimBlock2d>>>(devRGB, devHSV, devOutput, regionSize, inputImage->width, inputImage->height);
    
    hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
    
    // Free the device
    hipFree(devRGB);
    hipFree(devHSV.h);
    hipFree(devHSV.s);
    hipFree(devHSV.v);
    hipFree(devOutput);
    
    // Show the error
    r = hipGetLastError();
    if ( hipSuccess != r ) {
      printf("ERROR : %s\n", hipGetErrorString(r));
    }
}